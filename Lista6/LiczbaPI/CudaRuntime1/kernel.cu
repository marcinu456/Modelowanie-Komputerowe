﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>

#include <random>
#include <iomanip> 

int main(int argc, char* argv[]) {



    std::random_device rd{};
    std::mt19937 engine{ rd() };
    std::uniform_real_distribution<double> dis(0.0, DBL_MAX);
    long long N = 1'000'000'000;
    long long i;
    double x, y, n = 0, recv;

    hipMallocManaged(&x);

    for (i = 1; i < N; i++) {
        x = (dis(rd) / (DBL_MAX));
        y = (dis(rd) / (DBL_MAX));
        if (x * x + y * y <= 1) n++;
    }

    std::cout << "PI  = " << std::setprecision(10) << 4.0 * n / N << "\n";

    return 0;
}
